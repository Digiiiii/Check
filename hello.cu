
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello World from GPU thread %d!\n", threadIdx.x);
}

int main() {
    helloFromGPU<<<1, 5>>>();

    // Check for errors during kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Wait for kernel to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA device sync error: %s\n", hipGetErrorString(err));
        return -1;
    }

    printf("Hello World from CPU!\n");
    return 0;
}
